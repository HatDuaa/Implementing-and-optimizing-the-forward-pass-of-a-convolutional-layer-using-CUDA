#include "hip/hip_runtime.h"
#include "./conv_forward.h"
#include <cmath>
#include <iostream>

#define TILE_WIDTH_C1 16
#define TILE_WIDTH_C3 12

__constant__ float dc_filter[2400];


__global__ void conv_forward_kernel(float *out, const float *in, const int num_samples,
                                    const int num_output_channels, const int num_input_channels,
                                    const int h_in, const int w_in, const int filter_size)
{
    extern __shared__ float shared_input[];

    int TILE_WIDTH;    
    if (num_input_channels == 1){
        TILE_WIDTH = TILE_WIDTH_C1;
    }
    else{
        TILE_WIDTH = TILE_WIDTH_C3;
    }
    
    const int INPUT_TILE = TILE_WIDTH + filter_size - 1;
    
    const int h_out = h_in - filter_size + 1;
    const int w_out = w_in - filter_size + 1;


    #define o4d(i3, i2, i1, i0) out[(i3) * (num_output_channels * h_out * w_out) + (i2) * (h_out * w_out) + (i1) * (w_out) + i0]
    #define i4d(i3, i2, i1, i0) in[(i3) * (num_input_channels * h_in * w_in) + (i2) * (h_in * w_in) + (i1) * (w_in) + i0]
    #define f4d(i3, i2, i1, i0) dc_filter[(i3) * (num_input_channels * filter_size * filter_size) + (i2) * (filter_size * filter_size) + (i1) * (filter_size) + i0]
    #define sm3d(i2, i1, i0) shared_input[(i2) * (INPUT_TILE * INPUT_TILE) + (i1) * INPUT_TILE + i0]

    int w_grid = ceil(1.0*w_out / TILE_WIDTH); 
    int i_batch = blockIdx.x;                 // batch number
    int i_out_channel = blockIdx.y;           // output feature
    
    int ty = threadIdx.y;              // thread ID in the current TILE  
    int tx = threadIdx.x;
    
    int row_in = (blockIdx.z / w_grid) * TILE_WIDTH; // row of the input image matrix
    int col_in = (blockIdx.z % w_grid) * TILE_WIDTH; // col of the input image matrix
    
    int row_out = (blockIdx.z / w_grid) * TILE_WIDTH + ty; // row of the output image matrix
    int col_out = (blockIdx.z % w_grid) * TILE_WIDTH + tx; // col of the ouput image matrix    

    #pragma unroll
    for (int input_channel_idx = 0; input_channel_idx < num_input_channels; input_channel_idx++)
    {
        #pragma unroll
        for(int i = ty; i < INPUT_TILE; i += TILE_WIDTH)
        {
            #pragma unroll
            for(int j = tx; j < INPUT_TILE; j += TILE_WIDTH)
            {
                if (row_in + i < h_in && col_in + j < w_in)
                {
                    sm3d(input_channel_idx, i, j) = i4d(i_batch, input_channel_idx, row_in + i, col_in + j);
                }
            }
        }
    }

    // Make sure all threads loaded data into shared memory
    __syncthreads();

    // compute only within bounds
    if ((row_out < h_out) && (col_out < w_out)) 
    {
        float sum = 0.0f;
        #pragma unroll
        for(int input_channel_idx = 0; input_channel_idx < num_input_channels; input_channel_idx++)             // sum over all input features
        {
            #pragma unroll
            for(int p=0; p< filter_size; p++)         // KxK filter 
                #pragma unroll
                for(int q=0; q< filter_size; q++)
                    sum += sm3d(input_channel_idx, p + ty, q+tx) * f4d(i_out_channel, input_channel_idx, p, q); 
        }
        o4d(i_batch,i_out_channel,row_out,col_out) = sum;
    } 
    
    #undef sm3d
    #undef o4d
    #undef i4d
    #undef f4d
}

__host__ void GPUInterface::conv_forward_gpu(float *output_data, const float *input_data, const float *weight_data,
                                                  const int num_samples, const int output_channel, const int num_input_channels,
                                                  const int height_in, const int width_in, const int kernel_height)
{
    // Set the tile width
    int TILE_WIDTH;    
    if (num_input_channels == 1){
        TILE_WIDTH = TILE_WIDTH_C1;
    }
    else{
        TILE_WIDTH = TILE_WIDTH_C3;
    }
    

    // Allocate memory and copy over the relevant data structures to the GPU
    const int h_out = height_in - kernel_height + 1;
    const int w_out = width_in - kernel_height + 1;

    int inputSize = num_samples * num_input_channels * height_in * width_in * sizeof(float);
    int outputSize = num_samples * output_channel * h_out * w_out * sizeof(float);
    int filter_size = output_channel * num_input_channels * kernel_height * kernel_height * sizeof(float);

    float *device_input, *device_output, *device_kernel;

    hipMalloc((void **)&device_input, inputSize);
    hipMalloc((void **)&device_output, outputSize);
    hipMalloc((void **)&device_kernel, filter_size);

    hipMemcpy(device_input, input_data, inputSize, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(dc_filter), weight_data, filter_size);

    dim3 numThreadsPerBlock, numBlocksInGrid;

    int Z = ceil(1.0 * h_out / TILE_WIDTH) * ceil(1.0 * w_out / TILE_WIDTH);
    numThreadsPerBlock = dim3(TILE_WIDTH, TILE_WIDTH, 1);
    int shmem_size = num_input_channels * (TILE_WIDTH + kernel_height - 1) * (TILE_WIDTH + kernel_height - 1) * sizeof(float);
    numBlocksInGrid = dim3(num_samples, output_channel, Z);
    
    std::cout<<"\nGPU custom version 4:\n";
    // Launch the kernel
    conv_forward_kernel<<<numBlocksInGrid, numThreadsPerBlock, shmem_size>>>(device_output, device_input, num_samples, output_channel, num_input_channels, height_in, width_in, kernel_height);

    // Copy the output back to host
    hipMemcpy(output_data, device_output, outputSize, hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_kernel);

}
