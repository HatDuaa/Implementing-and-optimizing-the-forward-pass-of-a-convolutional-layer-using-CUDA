#include "hip/hip_runtime.h"
#include "./conv_forward.h"

#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float *out, const float *in, const float *filter,
    const int num_samples, const int num_output_channels, const int num_input_channels,
    const int h_in, const int w_in, const int filter_size)
{
    const int w_out = w_in - filter_size + 1;
    const int h_out = h_in - filter_size + 1;

    int width_grid = ceil(1.0 * w_out / TILE_WIDTH);
    int height_grid = ceil(1.0 * h_out / TILE_WIDTH);

    int i_batch = blockIdx.x;        // batch number
    int output_channel_idx = blockIdx.y; // output channel
    int row = (blockIdx.z / width_grid) * TILE_WIDTH + threadIdx.y; // row of the output matrix
    int col = (blockIdx.z % width_grid) * TILE_WIDTH + threadIdx.x; // column of the output matrix

    #define o4d(i3, i2, i1, i0) out[(i3) * (num_output_channels * h_out * w_out) + (i2) * (h_out * w_out) + (i1) * (w_out) + i0]
    #define i4d(i3, i2, i1, i0) in[(i3) * (num_input_channels * h_in * w_in) + (i2) * (h_in * w_in) + (i1) * (w_in) + i0]
    #define f4d(i3, i2, i1, i0) filter[(i3) * (num_input_channels * filter_size * filter_size) + (i2) * (filter_size * filter_size) + (i1) * (filter_size) + i0]


    if (row < h_out && col < w_out) 
    {
        float sum = 0.0f;
        for(int input_channel_idx = 0; input_channel_idx < num_input_channels; input_channel_idx++) // sum over all input channels
        {
            for(int filter_row = 0; filter_row < filter_size; filter_row++)  // filter_size x filter_size filter 
            {
                for(int filter_col = 0; filter_col < filter_size; filter_col++)
                {
                    int i_row = row + filter_row;
                    int i_col = col + filter_col;
                    // Tensor đầu vào có dạng: (num_samples, num_input_channels, h_in, w_in)
                    sum += i4d(i_batch, input_channel_idx, i_row, i_col) * 
                                f4d(output_channel_idx, input_channel_idx, filter_row, filter_col);
                }
            }
        }
        // Tensor đầu ra có dạng: (num_samples, num_output_channels, h_out, w_out)
        o4d(i_batch, output_channel_idx, row, col) = sum;
    }
    #undef o4d
    #undef i4d
    #undef f4d
}



__host__ void GPUInterface::conv_forward_gpu(float *output, const float *input, const float *filter,
    const int num_samples, const int num_output_channels, const int num_input_channels,
    const int h_in, const int w_in, const int filter_size)
{
    const int h_out = h_in - filter_size + 1;
    const int w_out = w_in - filter_size + 1;

    // Allocate device memory
    float *device_input, *device_output, *device_filter;
    hipMalloc((void **)&device_input, num_samples * num_input_channels * h_in * w_in * sizeof(float));  // input features map is num_input_channels
    hipMalloc((void **)&device_output, num_samples * num_output_channels * h_out * w_out * sizeof(float));  // output feature map is num_output_channels
    hipMalloc((void **)&device_filter, num_output_channels * num_input_channels * filter_size * filter_size * sizeof(float));  // num_input_channels * num_output_channels filter Maps of size filter_size * filter_size

    // Copy input and filter data to device
    hipMemcpy(device_input, input, num_samples * num_input_channels * h_in * w_in * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_filter, filter, num_output_channels * num_input_channels * filter_size * filter_size * sizeof(float), hipMemcpyHostToDevice);

    // Set the kernel dimensions and call the kernel
    int Z = ceil(1.0 * h_out / TILE_WIDTH) * ceil(1.0 * w_out / TILE_WIDTH);
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridSize(num_samples, num_output_channels, Z);

    std::cout<<"\nGPU custom version 1:\n";
    // Launch the kernel
    conv_forward_kernel<<<gridSize, blockSize>>>(device_output, device_input, device_filter, num_samples, num_output_channels, num_input_channels, h_in, w_in, filter_size);

    // Copy the output back to host
    hipMemcpy(output, device_output, num_samples * num_output_channels * h_out * w_out * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_filter);
}

